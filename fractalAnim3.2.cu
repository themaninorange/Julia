#include "hip/hip_runtime.h"
//I used to have make information up here.

#include <GL/glut.h>
#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <signal.h>

//#include <thrust/system_error.h>
//#include <thrust/system/cuda/error.h>
//#include <sstream>

#define PI 3.141592653589793238

//All includes which require glib
//This section is required because of a noinline conflict with nvcc
#undef __noinline__
#include <gtk/gtk.h>
#define __noinline__ __attribute__((noinline))

using namespace std;

float *pixelsJulia_CPU, *pixelsMandel_CPU, *pixels_CPU;

float *pixelsJulia_GPU, *pixelsMandel_GPU, *pixels_GPU;

//dim3 dimBlock;

//See coordinates.pdf for explanation of this coordinate system.
float m[] = {0, 0.75, 0, 0};
float c[] = {-2, -2, 0, 0};
float v[] = {4, 0, 0, 0};
float o[] = {0, 0, 0, 0};
float coordinates[] = {0,0};
//See coordinates.pdf for explanation of this coordinate system.

//GPU vectors for the coordinate system (to be copied later)
float *m_gpu, *c_gpu, *v_gpu, *o_gpu;

float t = 0;
float tmod = 100.0;
float titer = 1.0;
float moveiter = 0.001;

float viter = 0.001;

int N = 100;
int mandelID; 
int juliaID; 
unsigned int window_height = 960;
unsigned int window_width = 2*window_height;

int framecount = 0;
/*
float xMin = -2.0;
float xMax =  2.0;
float yMin = -2.0;
float yMax =  2.0;

float stepSizeX = (xMax - xMin)/((float)window_width);
float stepSizeY = (yMax - yMin)/((float)window_height);
*/

void ortho_coordinates(){

    //Given the view corner, view vector, and lat/long,
    //  return the 4 coordinates of the orthogonal vector.
    
    //Create a basis for the orthogonal complement of v.
    float e0[] = {-v[1], v[0],   0,   0};
    float e1[] = {-v[2],   0, v[0],   0};
    float e2[] = {-v[3],   0,   0, v[0]};
    
    //Length of v
    float R = sqrt(v[0]*v[0]+ v[1]*v[1]+ v[2]*v[2]+ v[3]*v[3]);
    
    //x,y,z coordinates given our lat/long coordinates
    float x = R*cos(coordinates[0])*cos(coordinates[1]);
    float y = R*cos(coordinates[0])*sin(coordinates[1]);
    float z = R*sin(coordinates[0]);
    
    //Transform the coordinates from the basis we just created to the overall space. 
    o[0] = x*e0[0] + y*e1[0] + z*e2[0];
    o[1] = x*e0[1] + y*e1[1] + z*e2[1];
    o[2] = x*e0[2] + y*e1[2] + z*e2[2];
    o[3] = x*e0[3] + y*e1[3] + z*e2[3];
    
    //length of o
    float r = sqrt(o[0]*o[0] + o[1]*o[1] + o[2]*o[2] + o[3]*o[3]);

    //Normalize o to the length of v.
    o[0] = R*o[0]/r;
    o[1] = R*o[1]/r;
    o[2] = R*o[2]/r;
    o[3] = R*o[3]/r;
}

static void
print_hello (GtkWidget *widget,
             gpointer   data)
{
  g_print ("Hello World\n");
}

static gboolean
on_delete_event (GtkWidget *widget,
                 GdkEvent  *event,
                 gpointer   data)
{
  /* If you return FALSE in the "delete_event" signal handler,
   * GTK will emit the "destroy" signal. Returning TRUE means
   * you don't want the window to be destroyed.
   *
   * This is useful for popping up 'are you sure you want to quit?'
   * type dialogs.
   */

  g_print ("delete event occurred\n");

  return TRUE;
}

void AllocateMemory(){
	hipMalloc((void**)&pixelsJulia_GPU, window_width/2*window_height*3*sizeof(float));
	pixelsJulia_CPU = (float *)malloc(window_width/2*window_height*3*sizeof(float));
	hipMalloc((void**)&pixelsMandel_GPU, window_width/2*window_height*3*sizeof(float));
	pixelsMandel_CPU = (float *)malloc(window_width/2*window_height*3*sizeof(float));
	hipMalloc((void**)&pixels_GPU, window_width*window_height*3*sizeof(float));
	pixels_CPU = (float *)malloc(window_width*window_height*3*sizeof(float));
	hipMalloc((void**)&m_gpu, 4*sizeof(float));
	hipMalloc((void**)&c_gpu, 4*sizeof(float));
	hipMalloc((void**)&v_gpu, 4*sizeof(float));
	hipMalloc((void**)&o_gpu, 4*sizeof(float));

} // */	//Saves the appropriate memory chunks for later use.
	//References the globally defined variables.


__global__ void cudaWeave(float *pixelsMandel_GPU, float *pixelsJulia_GPU, float *pixels_GPU){
	
	//red
	pixels_GPU[(2*blockIdx.x*blockDim.x + threadIdx.x)*3] = 
		pixelsJulia_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3];
		//First 600 on each row should be from the Julia set.
	
	pixels_GPU[((2*blockIdx.x+1)*blockDim.x + threadIdx.x)*3] = 
		pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3];	
		//601-1200 on each row should be from the Mandelbrot set.

	//green
	pixels_GPU[(2*blockIdx.x*blockDim.x + threadIdx.x)*3+1] = 
		pixelsJulia_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3+1];
		//""
	pixels_GPU[((2*blockIdx.x+1)*blockDim.x + threadIdx.x)*3+1] = 
		pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3+1];	
		//""	
	//blue
	pixels_GPU[(2*blockIdx.x*blockDim.x + threadIdx.x)*3+2] = 
		pixelsJulia_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3+2];
		//""
	pixels_GPU[((2*blockIdx.x+1)*blockDim.x + threadIdx.x)*3+2] = 
		pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3+2];	
		//""
}


__global__ void cudaColorJulia(float *pixelsJulia_GPU, float *m, float *c, float *v, float *o){
	
	//j = {1, i, j, k}
	//Start from the value for the first corner, and add values from v and o
	//  based on grid position.
	float j[] = {
	    c[0] + (((float)threadIdx.x)/(blockDim.x))*v[0] + (((float)blockIdx.x)/(gridDim.x))*o[0],
	    c[1] + (((float)threadIdx.x)/(blockDim.x))*v[1] + (((float)blockIdx.x)/(gridDim.x))*o[1],
	    c[2] + (((float)threadIdx.x)/(blockDim.x))*v[2] + (((float)blockIdx.x)/(gridDim.x))*o[2],
	    c[3] + (((float)threadIdx.x)/(blockDim.x))*v[3] + (((float)blockIdx.x)/(gridDim.x))*o[3],
	};
	
	float mag,maxMag, t;
	int maxCount = 200;
	int count = 0;
	maxMag = 10;
	mag = 0.0;

	while (mag < maxMag && count < maxCount) 
	{
		t = j[0];
			
		j[0] = j[0]*j[0] - j[1]*j[1] - j[2]*j[2] - j[3]*j[3] + m[0];
		j[1] = 2*t*j[1]+m[1];
		j[2] = 2*t*j[2]+m[2];
		j[3] = 2*t*j[3]+m[3];
		mag = sqrt(j[0]*j[0] + j[1]*j[1] + j[2]*j[2] + j[3]*j[3]);
		count++;
	}
	if(count < maxCount) 
	{
		pixelsJulia_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3] = 
			0.5*log((double)count)/log((double)maxCount); 
		pixelsJulia_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 1] =
			1.0*log((double)count)/log((double)maxCount); 
		pixelsJulia_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 2] = 
			0.4;
	}
	else
	{
		pixelsJulia_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3] = 0.0;
		pixelsJulia_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 1] = 0.0;
		pixelsJulia_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 2] = 0.0;

	}
	

}

__global__ void cudaColorMandelbrot(float *pixelsMandel_GPU, float xseed, float yseed){
	
	float x = 0;
	float y = 0;
	float X = (((float)threadIdx.x)/(blockDim.x))*4-2.5;
	float iY = (((float)blockIdx.x)/(gridDim.x))*4-2;
	float mag,maxMag, t1;
	int maxCount = 200;
	int count = 0;
	maxMag = 10;
	mag = 0.0;
	
	if ((abs(xseed - (float)threadIdx.x/blockDim.x*4+2.5) <= 2.0/blockDim.x) && (abs(yseed - (float)blockIdx.x/gridDim.x*4+2)) <=2.0/gridDim.x){
	//If this pixel corresponds to the seed for the Julia set being generated,
		pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3] = 1.0;
		pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 1] = 0.0;
		pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 2] = 0.0;
		//... make this pixel red.
	}
	else{
	//Otherwise, find the color the way we normally would with the Mandelbrot set.
	
		while (mag < maxMag && count < maxCount){
		//As long as the complex number doesn't get farther than a certain distance,
		// and as long as we haven't iterated this enough times,
			t1 = x;	
			x = x*x - y*y + X;
			y = (2.0 * t1 * y) + iY;
			mag = sqrt(x*x + y*y);
			count++;
			//... find the next point in the sequence and update to it.
		}
		if(count < maxCount){
		//If we broke the above loop before iterating as many times as we want,
		// then the sequence diverges,
		
			pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3] = 
				0.5*log((double)count)/log((double)maxCount); 
			pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 1] =
				1.0*log((double)count)/log((double)maxCount); 
			pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 2] = 
				0.4;
			//... and we color it prettily according to how quickly it diverged.
		}
		else
		//Otherwise, the point is in the mandelbrot set (or close enough to it),
		{
			pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3] = 0.0;
			pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 1] = 0.0;
			pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 2] = 0.0;
			//... and we color it black.
		}
	}
}
void update(int value){
	
	//t = t + titer;
	/*A = -pow(sin(t/tmod),2);
	B = sin(2*t/tmod)/2;// */
	/*A = -pow(sin((2*t/5)/50.0),2);
	B = sin((2*t/3)/50.0)/2;// */
	/*A = -pow(sin((2.0*t/5)/tmod),2);
	B = sin((sqrt(5)+1)/2*(t)/tmod)/2;// */
	glutPostRedisplay();
	glutTimerFunc(16,update, 0);
}

void processSpecialKeys(int key, int x, int y) {

	switch(key) {
		case GLUT_KEY_RIGHT :
			m[0] = m[0] + moveiter;	break;
		case GLUT_KEY_LEFT :
			m[0] = m[0] - moveiter;	break;
		case GLUT_KEY_UP :
			m[1] = m[1] + moveiter;	break;
		case GLUT_KEY_DOWN :
			m[1] = m[1] - moveiter;	break;
	}
	
}

void view_zoom_in() {

    //The corner moves slightly in the direction of v and o.
    //Then v becomes smaller and o is recalculated.
    
    c[0] = c[0] + viter * v[0] + viter * o[0];
    c[1] = c[1] + viter * v[1] + viter * o[1];
    c[2] = c[2] + viter * v[2] + viter * o[2];
    c[3] = c[3] + viter * v[3] + viter * o[3];
    
    v[0] = v[0] - 2*viter*v[0];
    v[1] = v[1] - 2*viter*v[1];
    v[2] = v[2] - 2*viter*v[2];
    v[3] = v[3] - 2*viter*v[3];
        
    ortho_coordinates();
    
}

void view_zoom_out() {
    
    //The corner moves slightly in the OPPOSITE direction of v and o.
    //Then v becomes larger and o is recalculated.
    
    c[0] = c[0] - viter * v[0] - viter * o[0];
    c[1] = c[1] - viter * v[1] - viter * o[1];
    c[2] = c[2] - viter * v[2] - viter * o[2];
    c[3] = c[3] - viter * v[3] - viter * o[3];
    
    v[0] = v[0] + 2*viter*v[0];
    v[1] = v[1] + 2*viter*v[1];
    v[2] = v[2] + 2*viter*v[2];
    v[3] = v[3] + 2*viter*v[3];
        
    ortho_coordinates();
    
}

void processNormalKeys(unsigned char key, int x, int y) {
    
    switch(key) {
        case 49: // 1
        case 33: // !
		    c[0] = c[0] + viter;   break;
        case 50: // 2
        case 64: // @
		    c[1] = c[1] + viter;   break;
        case 51: // 3
        case 35: // #
		    c[2] = c[2] + viter;   break;
        case 52: // 4
        case 36: // $
		    c[3] = c[3] + viter;   break;
        case 53: // 5
        case 37: // %
		    v[0] = v[0] + viter;   break;
        case 54: // 6
        case 94: // ^
		    v[1] = v[1] + viter;   break;
        case 55: // 7
        case 38: // &
		    v[2] = v[2] + viter;   break;
        case 56: // 8
        case 42: // *
		    v[3] = v[3] + viter;   break;
        case 57: // 9
        case 40: // (
		    coordinates[0] = min(coordinates[0] + viter, PI/2);   break;
        case 48: // 0
        case 41: // )
            coordinates[1] = coordinates[1] + viter > 2*PI ? 0 : coordinates[1] + viter;   break;
        case 45: // -
        case 95: // _
		    viter = viter * 1.2;  break; 
        case 61: // =
        case 43: // +
		    view_zoom_in();   break;
        case 81: // Q
        case 113: // q
		    c[0] = c[0] - viter;   break; 
        case 87: // W
        case 119: // w
		    c[1] = c[1] - viter;   break; 
        case 69: // E
        case 101: // e
		    c[2] = c[2] - viter;   break; 
        case 82: // R
        case 114: // r
		    c[3] = c[3] - viter;   break; 
        case 84: // T
        case 116: // t
		    v[0] = v[0] - viter;   break; 
        case 89: // Y
        case 121: // y
		    v[1] = v[1] - viter;   break; 
        case 85: // U
        case 117: // u
		    v[2] = v[2] - viter;   break; 
        case 73: // I
        case 105: // i
		    v[3] = v[3] - viter;   break; 
        case 79: // O
        case 111: // o
		    coordinates[0] = max(coordinates[0] - viter, -PI/2);   break;
        case 80: // P
        case 112: // p
            coordinates[1] = coordinates[1] - viter < 0 ? 2*PI : coordinates[1] - viter;   break;
        case 91: // [
        case 123: // {
		    viter = viter / 1.2;  break; 
        case 93: // ]
        case 125: // }
		    view_zoom_out();   break; 
        case 65: // A
        case 97: // a
		    m[0] = m[0] + moveiter;   break; 
        case 83: // S
        case 115: // s
		    m[1] = m[1] + moveiter;   break; 
        case 68: // D
        case 100: // d
		    m[2] = m[2] + moveiter;   break; 
        case 70: // F
        case 102: // f
		    m[3] = m[3] + moveiter;   break; 
        case 90: // Z
        case 122: // z
		    m[0] = m[0] - moveiter;   break; 
        case 88: // X
        case 120: // x
		    m[1] = m[1] - moveiter;   break; 
        case 67: // C
        case 99: // c
		    m[2] = m[2] - moveiter;   break; 
        case 86: // V
        case 118: // v
		    m[3] = m[3] - moveiter;   break; 
        case 71: // G
        case 103: // g
		    moveiter = moveiter * 1.2;  break;
        case 66: // B
        case 98: // b
		    moveiter = moveiter/1.2;    break;
	}
}

void mouseClicks(int button, int state, int x, int y) {

	/*switch(button) {
		case GLUT_LEFT_BUTTON :
			A = ((float)x-window_width/2)/window_width*2.0-2.5;
			B = -(float)y/window_height*2.0-2.0;	break;
		case GLUT_RIGHT_BUTTON :
			break;
	}*/
	
	switch(button) {
		case GLUT_LEFT_BUTTON :
			m[0] = ((float)x)/window_width*8.0-6.5;
			m[1] = (-(float)y)/window_height*4.0+2.0;	break;
		case GLUT_RIGHT_BUTTON :
			break;
	}

	
}// */

/*void displayJulia(void) 
{ 
	glutSetWindow(juliaID);
	cudaColorJulia<<<window_width, window_height>>>(pixelsJulia_GPU, A, B);
	hipMemcpy(pixelsJulia_CPU, pixelsJulia_GPU,
		window_width*window_height*3*sizeof(float),
		hipMemcpyDeviceToHost);
	glDrawPixels(window_width, window_height, GL_RGB, GL_FLOAT, pixelsJulia_CPU); 
	glFlush(); 
	
}

void displayMandelbrot(void) 
{
	glutSetWindow(mandelID);
	cudaColorMandelbrot<<<window_width, window_height>>>(pixelsMandel_GPU, A, B);
	hipMemcpy(pixelsMandel_CPU, pixelsMandel_GPU,
		window_width*window_height*3*sizeof(float),
		hipMemcpyDeviceToHost);
	glDrawPixels(window_width, window_height, GL_RGB, GL_FLOAT, pixelsMandel_CPU); 
	glFlush(); 
}// */


void weavePixels(){
	
	hipMemcpy(pixelsJulia_GPU, pixelsJulia_CPU,
		window_width/2*window_height*3*sizeof(float),
		hipMemcpyHostToDevice);
	hipMemcpy(pixelsMandel_GPU, pixelsMandel_CPU,
		window_width/2*window_height*3*sizeof(float),
		hipMemcpyHostToDevice);
	
	cudaWeave<<<window_width/2,window_height>>>(pixelsMandel_GPU, pixelsJulia_GPU, pixels_GPU);

	hipMemcpy(pixels_CPU, pixels_GPU,
		window_width*window_height*3*sizeof(float),
		hipMemcpyDeviceToHost);

}// */

/*void throw_on_cuda_error(hipError_t code, const char *file, int line)
{
  if(code != hipSuccess)
  {
    std::stringstream ss;
    ss << file << "(" << line << ")";
    std::string file_and_line;
    ss >> file_and_line;
    throw thrust::system_error(code, thrust::cuda_category(), file_and_line);
  }
}*/

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void display(void){

    framecount = framecount + 1;
    
    ortho_coordinates();
    
    if (framecount%100 == 0){
        printf("frame: %d \n", framecount);
        printf("m: %f, %f *i, %f *j, %f *k \n" , m[0], m[1], m[2], m[3]);
        printf("c: %f, %f *i, %f *j, %f *k \n" , c[0], c[1], c[2], c[3]);
        printf("v: %f, %f *i, %f *j, %f *k \n" , v[0], v[1], v[2], v[3]);
        printf("o: %f, %f *i, %f *j, %f *k \n" , o[0], o[1], o[2], o[3]);
    };
    hipMemcpy(m_gpu, m, 4*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c_gpu, c, 4*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(v_gpu, v, 4*sizeof(float), hipMemcpyHostToDevice);
    gpuErrchk(hipMemcpy(o_gpu, o, 4*sizeof(float), hipMemcpyHostToDevice));
    
    
	cudaColorJulia<<<window_width/2, window_height>>>(pixelsJulia_GPU, m_gpu, c_gpu, v_gpu, o_gpu);
	hipMemcpy(pixelsJulia_CPU, pixelsJulia_GPU,
		window_width/2*window_height*3*sizeof(float),
		hipMemcpyDeviceToHost);
	cudaColorMandelbrot<<<window_width/2, window_height>>>(pixelsMandel_GPU, m[0], m[1]);
	hipMemcpy(pixelsMandel_CPU, pixelsMandel_GPU,
		window_width/2*window_height*3*sizeof(float),
		hipMemcpyDeviceToHost);

	weavePixels();

/*	//glRasterPos2i(0,0);
	glDrawPixels(window_width/2, window_height, GL_RGB, GL_FLOAT, pixelsJulia_CPU);
	//glRasterPos2i(window_width/2,0);
	glDrawPixels(window_width/2, window_height, GL_RGB, GL_FLOAT, pixelsMandel_CPU); // */

	glDrawPixels(window_width, window_height, GL_RGB, GL_FLOAT, pixels_CPU);
    /*
    glFontBegin(&font);
    glScalef(8.0, 8.0, 8.0);
    glTranslatef(30, 30, 0);
    glFontTextOut("Test", 5, 5, 0);
    glFontEnd();
    */
	glFlush(); 


}

void CleanUp(float *A_CPU, float *B_CPU, float *C_CPU, 
	     float *A_GPU, float *B_GPU, float *C_GPU){
	free(A_CPU);
	free(B_CPU);
	free(C_CPU);

	hipFree(A_GPU);
	hipFree(B_GPU);
	hipFree(C_GPU);
} // */	


int main(int argc, char *argv[])
{ 
	if(argc == 2){
		char *ptr;
		N = strtol(argv[1], &ptr, 10);
	}
	else if(argc > 2){
		printf("One or zero arguments expected.");
		return(1);
	}
	
	AllocateMemory();
	
   	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
   	glutInitWindowSize(window_width, window_height);
   	
   	/*glutCreateWindow("Subwindow");
   	glutDisplayFunc(displayMandelbrot);
   	mandelID = glutGetWindow();
   	glutMouseFunc(mouseClicks);
	glutSpecialFunc(processSpecialKeys);
	glutKeyboardFunc(processNormalKeys);// */
	
   	glutCreateWindow("Fractals man, fractals.");
   	glutDisplayFunc(display);
   	//juliaID = glutGetWindow();
   	glutMouseFunc(mouseClicks);
	//glutSpecialFunc(processSpecialKeys);
	glutKeyboardFunc(processNormalKeys);
	glutSpecialFunc(processSpecialKeys);
	glutTimerFunc(16, update, 0);
	glutMainLoop();
}

