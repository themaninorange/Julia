#include "hip/hip_runtime.h"
//I used to have make information up here.

#include <GL/glut.h>
#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <signal.h>

using namespace std;

float *pixelsJulia_CPU, *pixelsMandel_CPU, *pixels_CPU;

float *pixelsJulia_GPU, *pixelsMandel_GPU, *pixels_GPU;

//dim3 dimBlock;


float A = 0;    //1 coordinate of seed
float B = 0.75; //i coordinate of seed
float C = 0;    //j coordinate of seed
float D = 0;    //k coordinate of seed

float vA = 0;    //1 coordinate of View
float vB = 0;    //i coordinate of View
float vC = 0;    //j coordinate of View
float vD = 0;    //k coordinate of View
float vE = 0;    //xi1 hopf coordinate of View
float vF = 0;    //xi2 hopf coordinate of View
float vG = 0;    //eta hopf coordinate of View

float t = 0;
float tmod = 100.0;
float titer = 1.0;
float moveiter = 0.001;
int N = 100;
int mandelID; 
int juliaID; 
unsigned int window_height = 960;
unsigned int window_width = 2*window_height;

/*
float xMin = -2.0;
float xMax =  2.0;
float yMin = -2.0;
float yMax =  2.0;

float stepSizeX = (xMax - xMin)/((float)window_width);
float stepSizeY = (yMax - yMin)/((float)window_height);
*/

void AllocateMemory(){
	hipMalloc((void**)&pixelsJulia_GPU, window_width/2*window_height*3*sizeof(float));
	pixelsJulia_CPU = (float *)malloc(window_width/2*window_height*3*sizeof(float));
	hipMalloc((void**)&pixelsMandel_GPU, window_width/2*window_height*3*sizeof(float));
	pixelsMandel_CPU = (float *)malloc(window_width/2*window_height*3*sizeof(float));
	hipMalloc((void**)&pixels_GPU, window_width*window_height*3*sizeof(float));
	pixels_CPU = (float *)malloc(window_width*window_height*3*sizeof(float));

} // */	//Saves the appropriate memory chunks for later use.
	//References the globally defined variables.

float color (float x, float y)	//hopefully centered on (0,0)? 
{
	float mag,maxMag,t1;
	float maxCount = 200;
	float count = 0;
	maxMag = 10;
	mag = 0.0;

	while (mag < maxMag && count < maxCount) 
	{
		t1 = x;	
		x = x*x - y*y + A;
		y = (2.0 * t1 * y) + B;
		mag = sqrt(x*x + y*y);
		count++;
	}
	if(count < maxCount) 
	{
		return(1.0);
	}
	else
	{
		return(0.0);
	}
}

__global__ void cudaWeave(float *pixelsMandel_GPU, float *pixelsJulia_GPU, float *pixels_GPU){
	
	//red
	pixels_GPU[(2*blockIdx.x*blockDim.x + threadIdx.x)*3] = 
		pixelsJulia_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3];
		//First 600 on each row should be from the Julia set.
	
	pixels_GPU[((2*blockIdx.x+1)*blockDim.x + threadIdx.x)*3] = 
		pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3];	
		//601-1200 on each row should be from the Mandelbrot set.

	//green
	pixels_GPU[(2*blockIdx.x*blockDim.x + threadIdx.x)*3+1] = 
		pixelsJulia_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3+1];
		//""
	pixels_GPU[((2*blockIdx.x+1)*blockDim.x + threadIdx.x)*3+1] = 
		pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3+1];	
		//""	
	//blue
	pixels_GPU[(2*blockIdx.x*blockDim.x + threadIdx.x)*3+2] = 
		pixelsJulia_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3+2];
		//""
	pixels_GPU[((2*blockIdx.x+1)*blockDim.x + threadIdx.x)*3+2] = 
		pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3+2];	
		//""
}


__global__ void cudaColorJulia(float *pixelsJulia_GPU, float A, float B, float C, float D){
	
	float a = (((float)threadIdx.x)/(blockDim.x))*4-2;
	float b = (((float)blockIdx.x)/(gridDim.x))*4-2;
	float c = 0;
	float d = 0;
	
	float mag,maxMag, ta;
	int maxCount = 200;
	int count = 0;
	maxMag = 10;
	mag = 0.0;

	while (mag < maxMag && count < maxCount) 
	{
		ta = a;
			
		a = a*a - b*b - c*c - d*d + A;
		b = 2*ta*b+B;
		c = 2*ta*c+C;
		d = 2*ta*d+D;
		mag = sqrt(a*a + b*b + c*c + d*d);
		count++;
	}
	if(count < maxCount) 
	{
		pixelsJulia_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3] = 
			0.5*log((double)count)/log((double)maxCount); 
		pixelsJulia_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 1] =
			1.0*log((double)count)/log((double)maxCount); 
		pixelsJulia_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 2] = 
			0.4;
	}
	else
	{
		pixelsJulia_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3] = 0.0;
		pixelsJulia_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 1] = 0.0;
		pixelsJulia_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 2] = 0.0;

	}
	

}

__global__ void cudaColorMandelbrot(float *pixelsMandel_GPU, float xseed, float yseed){
	
	float x = 0;
	float y = 0;
	float X = (((float)threadIdx.x)/(blockDim.x))*4-2.5;
	float iY = (((float)blockIdx.x)/(gridDim.x))*4-2;
	float mag,maxMag, t1;
	int maxCount = 200;
	int count = 0;
	maxMag = 10;
	mag = 0.0;
	
	if ((abs(xseed - (float)threadIdx.x/blockDim.x*4+2.5) <= 2.0/blockDim.x) && (abs(yseed - (float)blockIdx.x/gridDim.x*4+2)) <=2.0/gridDim.x){
	//If this pixel corresponds to the seed for the Julia set being generated,
		pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3] = 1.0;
		pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 1] = 0.0;
		pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 2] = 0.0;
		//... make this pixel red.
	}
	else{
	//Otherwise, find the color the way we normally would with the Mandelbrot set.
	
		while (mag < maxMag && count < maxCount){
		//As long as the complex number doesn't get farther than a certain distance,
		// and as long as we haven't iterated this enough times,
			t1 = x;	
			x = x*x - y*y + X;
			y = (2.0 * t1 * y) + iY;
			mag = sqrt(x*x + y*y);
			count++;
			//... find the next point in the sequence and update to it.
		}
		if(count < maxCount){
		//If we broke the above loop before iterating as many times as we want,
		// then the sequence diverges,
		
			pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3] = 
				0.5*log((double)count)/log((double)maxCount); 
			pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 1] =
				1.0*log((double)count)/log((double)maxCount); 
			pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 2] = 
				0.4;
			//... and we color it prettily according to how quickly it diverged.
		}
		else
		//Otherwise, the point is in the mandelbrot set (or close enough to it),
		{
			pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3] = 0.0;
			pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 1] = 0.0;
			pixelsMandel_GPU[(blockIdx.x*blockDim.x + threadIdx.x)*3 + 2] = 0.0;
			//... and we color it black.
		}
	}
}
void update(int value){
	
	//t = t + titer;
	/*A = -pow(sin(t/tmod),2);
	B = sin(2*t/tmod)/2;// */
	/*A = -pow(sin((2*t/5)/50.0),2);
	B = sin((2*t/3)/50.0)/2;// */
	/*A = -pow(sin((2.0*t/5)/tmod),2);
	B = sin((sqrt(5)+1)/2*(t)/tmod)/2;// */
	glutPostRedisplay();
	glutTimerFunc(16,update, 0);
}

/*static void signalHandler(int signum) {
	int command;
	bool exitMenu = 0;
	//cout << "I handled it :)" << endl;
	
	while (exitMenu == 0) {
	    	cout << "Enter 0 to exit the program." << endl;
		cout << "Enter 1 to continue." << endl;
		cin >> command;
		
		if(command == 0) {
			exitMenu = 1;
		} else if (command == 1){
			exitMenu = 1;
			cout << "resuming..." << endl;
		} else {
			cout << "Invalid Command!" << endl;
		}
		cout << endl;
	}
}// */

/*void processSpecialKeys(int key, int x, int y) {

	switch(key) {
		case GLUT_KEY_RIGHT :
				t = t + titer*100;	break;
		case GLUT_KEY_LEFT :
				t = t - titer*100;	break;
		case GLUT_KEY_UP :
				titer = titer*1.1;	break;
		case GLUT_KEY_DOWN :
				titer = titer/1.1;	break;
	}
}// */

void processSpecialKeys(int key, int x, int y) {

	switch(key) {
		case GLUT_KEY_RIGHT :
			A = A + moveiter;	break;
		case GLUT_KEY_LEFT :
			A = A - moveiter;	break;
		case GLUT_KEY_UP :
			B = B + moveiter;	break;
		case GLUT_KEY_DOWN :
			B = B - moveiter;	break;
	}
	
}

void processNormalKeys(unsigned char key, int x, int y) {
    
    switch(key) {
	    case 43:	//Plus sign key, '+'
		    moveiter = moveiter * 1.2;  break;
	    case 45:	//Minus sign key, '-'
		    moveiter = moveiter/1.2;    break;
	    case 81:	//Q = 17 + 64
	    case 113:	//q = 17 + 96
		    A = A + moveiter;   break;
	    case 65:	//A = 1  + 64
	    case 97:	//a = 1  + 96
		    A = A - moveiter;   break;
	    case 87:	//W = 23 + 64
	    case 119:	//w = 23 + 96
		    B = B + moveiter;   break;
	    case 83:	//S = 19 + 64
	    case 115:	//s = 19 + 96
		    B = B - moveiter;   break;
	    case 69:	//E = 5  + 64
	    case 101:	//e = 5  + 96
		    C = C + moveiter;   break;
	    case 68:	//D = 4  + 64
	    case 100:	//d = 4  + 96
		    C = C - moveiter;   break;
	    case 82:	//R = 18 + 64
	    case 114:	//r = 18 + 96
		    D = D + moveiter;   break;
	    case 70:	//F = 6  + 64
	    case 102:	//f = 6  + 96
		    D = D - moveiter;   break;
	    case 84:	//T = 20 + 64
	    case 116:	//t = 20 + 96
		    D = D - moveiter;   break;
	    case 71:	//G = 7  + 64
	    case 103:	//g = 7  + 96
		    D = D - moveiter;   break;
	    case 89:	//Y = 25 + 64
	    case 121:	//y = 25 + 96
		    D = D - moveiter;   break;
	    case 72:	//H = 8  + 64
	    case 104:	//h = 8  + 96
		    D = D - moveiter;   break;
	    case 85:	//U = 21 + 64
	    case 117:	//u = 21 + 96
		    D = D - moveiter;   break;
	    case 74:	//J = 10 + 64
	    case 106:	//j = 10 + 96
		    D = D - moveiter;   break;
	    case 73:	//I = 9  + 64
	    case 105:	//i = 9  + 96
		    D = D - moveiter;   break;
	    case 75:	//K = 11 + 64
	    case 107:	//k = 11 + 96
		    D = D - moveiter;   break;
	    case 79:	//O = 15 + 64
	    case 111:	//o = 15 + 96
		    D = D - moveiter;   break;
	    case 76:	//L = 12 + 64
	    case 108:	//l = 12 + 96
		    D = D - moveiter;   break;
	    case 80:	//P = 16 + 64
	    case 112:	//p = 16 + 96
		    D = D - moveiter;   break;
	    case 59:	//; = 59
	    case 58:	//: = 58
		    D = D - moveiter;   break;
	    case 91:	//[ = 91
	    case 123:	//{ = 123
		    D = D - moveiter;   break;
	    case 44:	//' = 44
	    case 34:	//" = 34
		    D = D - moveiter;   break;
	}
}

void mouseClicks(int button, int state, int x, int y) {

	/*switch(button) {
		case GLUT_LEFT_BUTTON :
			A = ((float)x-window_width/2)/window_width*2.0-2.5;
			B = -(float)y/window_height*2.0-2.0;	break;
		case GLUT_RIGHT_BUTTON :
			break;
	}*/
	
	switch(button) {
		case GLUT_LEFT_BUTTON :
			A = ((float)x)/window_width*8.0-6.5;
			B = (-(float)y)/window_height*4.0+2.0;	break;
		case GLUT_RIGHT_BUTTON :
			break;
	}

	
}// */

/*void displayJulia(void) 
{ 
	glutSetWindow(juliaID);
	cudaColorJulia<<<window_width, window_height>>>(pixelsJulia_GPU, A, B);
	hipMemcpy(pixelsJulia_CPU, pixelsJulia_GPU,
		window_width*window_height*3*sizeof(float),
		hipMemcpyDeviceToHost);
	glDrawPixels(window_width, window_height, GL_RGB, GL_FLOAT, pixelsJulia_CPU); 
	glFlush(); 
	
}

void displayMandelbrot(void) 
{
	glutSetWindow(mandelID);
	cudaColorMandelbrot<<<window_width, window_height>>>(pixelsMandel_GPU, A, B);
	hipMemcpy(pixelsMandel_CPU, pixelsMandel_GPU,
		window_width*window_height*3*sizeof(float),
		hipMemcpyDeviceToHost);
	glDrawPixels(window_width, window_height, GL_RGB, GL_FLOAT, pixelsMandel_CPU); 
	glFlush(); 
}// */


void weavePixels(){
	
	hipMemcpy(pixelsJulia_GPU, pixelsJulia_CPU,
		window_width/2*window_height*3*sizeof(float),
		hipMemcpyHostToDevice);
	hipMemcpy(pixelsMandel_GPU, pixelsMandel_CPU,
		window_width/2*window_height*3*sizeof(float),
		hipMemcpyHostToDevice);
	
	cudaWeave<<<window_width/2,window_height>>>(pixelsMandel_GPU, pixelsJulia_GPU, pixels_GPU);

	hipMemcpy(pixels_CPU, pixels_GPU,
		window_width*window_height*3*sizeof(float),
		hipMemcpyDeviceToHost);

}// */

void display(void){

	cudaColorJulia<<<window_width/2, window_height>>>(pixelsJulia_GPU, A, B, C, D);
	hipMemcpy(pixelsJulia_CPU, pixelsJulia_GPU,
		window_width/2*window_height*3*sizeof(float),
		hipMemcpyDeviceToHost);
	cudaColorMandelbrot<<<window_width/2, window_height>>>(pixelsMandel_GPU, A, B);
	hipMemcpy(pixelsMandel_CPU, pixelsMandel_GPU,
		window_width/2*window_height*3*sizeof(float),
		hipMemcpyDeviceToHost);

	weavePixels();

/*	//glRasterPos2i(0,0);
	glDrawPixels(window_width/2, window_height, GL_RGB, GL_FLOAT, pixelsJulia_CPU);
	//glRasterPos2i(window_width/2,0);
	glDrawPixels(window_width/2, window_height, GL_RGB, GL_FLOAT, pixelsMandel_CPU); // */

	glDrawPixels(window_width, window_height, GL_RGB, GL_FLOAT, pixels_CPU);
    /*
    glFontBegin(&font);
    glScalef(8.0, 8.0, 8.0);
    glTranslatef(30, 30, 0);
    glFontTextOut("Test", 5, 5, 0);
    glFontEnd();
    */
	glFlush(); 


}

void CleanUp(float *A_CPU, float *B_CPU, float *C_CPU, 
	     float *A_GPU, float *B_GPU, float *C_GPU){
	free(A_CPU);
	free(B_CPU);
	free(C_CPU);

	hipFree(A_GPU);
	hipFree(B_GPU);
	hipFree(C_GPU);
} // */	


int main(int argc, char *argv[])
{ 
	if(argc == 2){
		char *ptr;
		N = strtol(argv[1], &ptr, 10);
	}
	else if(argc > 2){
		printf("One or zero arguments expected.");
		return(1);
	}
	
	AllocateMemory();
	
   	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
   	glutInitWindowSize(window_width, window_height);
   	
   	/*glutCreateWindow("Subwindow");
   	glutDisplayFunc(displayMandelbrot);
   	mandelID = glutGetWindow();
   	glutMouseFunc(mouseClicks);
	glutSpecialFunc(processSpecialKeys);
	glutKeyboardFunc(processNormalKeys);// */
	
   	glutCreateWindow("Fractals man, fractals.");
   	glutDisplayFunc(display);
   	//juliaID = glutGetWindow();
   	glutMouseFunc(mouseClicks);
	//glutSpecialFunc(processSpecialKeys);
	glutKeyboardFunc(processNormalKeys);
	glutSpecialFunc(processSpecialKeys);
	glutTimerFunc(16, update, 0);
	glutMainLoop();
}

